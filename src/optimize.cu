#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <cmath>
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "optimize.cuh"

#include <hiprand.h>


using namespace std;


/* User defined objective function goes here. */
__device__ __host__ float f(float x) {
  return 20 * log(x) - .0004 * x * x * sin(x) + x * cos(x);
}

/* Generate a float between low and high, given r between 0 and 1. */
__device__ float rand_float(float low, float high, float r) {
  return low + r * (high - low);
}

/* Apply objective function to all inputs. */
__global__
void applyFunctionKernel(float *input, float *output, int N, float low, float high) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < N) {
    float r = input[index];

    /* Convert r (which is in [0, 1]) into a valid input and
     * write this to the input. This is convenient to do here
     */
    input[index] = rand_float(low, high, r);

    output[index] = f(rand_float(low, high, r));

    index += gridDim.x * blockDim.x;
  }
}

void callApplyFunctionKernel(float *input, float *output, int N, float low, float high) {

  int block_count = 32;
  int threads_per_block = 32;

  applyFunctionKernel<<<block_count, threads_per_block>>>(input, output, N, low, high);
}

/* A bit of boilerplate to allow for a critical section. Borrowed from
 * http://stackoverflow.com/questions/18963293/cuda-atomics-change-flag
 */

__device__ volatile int sem = 0;

__device__ void acquire_semaphore(volatile int *lock){
  while (atomicCAS((int *)lock, 0, 1) != 0);
  }

__device__ void release_semaphore(volatile int *lock){
  *lock = 0;
  __threadfence();
}

/* Find max of a subset by using shmem, then combine solutions
 * across blocks. */
__global__
void findMaxKernel(float *output, int N, float *max, int *max_index) {
  /* Local max of the values looked at by this block */
  __shared__ float local_max;
  __shared__ int local_max_index;

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  /* Initialize shmem */
  if (threadIdx.x == 0) {
    local_max = output[0];
    local_max_index = 0;
  }

  if (index == 0) {
    *max = output[0];
    *max_index = 0;
  }

  while (index < N) {

    float tmp = output[index];

    if (tmp > local_max) {
      local_max = tmp;
      local_max_index = index;
    }

    index += gridDim.x * blockDim.x;
  }

  __syncthreads();

  /* One thread per block compares values across blocks */

  if (threadIdx.x == 0) {
    __syncthreads();
    if (threadIdx.x == 0) {
      acquire_semaphore(&sem);
    }
    __syncthreads();

    // critical section
    if (local_max > *max) {
      *max = local_max;
      *max_index = local_max_index;
    }

    __syncthreads();
    if (threadIdx.x == 0) {
      release_semaphore(&sem);
    }
    __syncthreads();

  }
}

OptimizationOutput cudaCallMaximumKernel(float *input, float *output, int N) {
  
  int block_count = 32;
  int threads_per_block = 32;

  /* Set up buffers for GPU */

  float *dev_max;
  int *dev_max_index;

  gpuErrChk(hipMalloc(&dev_max, sizeof(float)));
  gpuErrChk(hipMalloc(&dev_max_index, sizeof(int)));

  /* Call kernel */
  findMaxKernel<<<block_count, threads_per_block>>>(output, N, dev_max,
    dev_max_index);

  /* Now get values off of GPU. This part is a bit awkward: need the
   * index in order to find the corresponding input */

  float *host_maximizer = (float *) malloc(sizeof(float));
  float *host_max = (float *) malloc(sizeof(float));
  int *host_max_index = (int *) malloc(sizeof(int));

  *host_maximizer = -1.;
  *host_max = -1.;
  *host_max_index = -1;

  gpuErrChk(hipMemcpy(host_max, dev_max,
    sizeof(float), hipMemcpyDeviceToHost));

  gpuErrChk(hipMemcpy(host_max_index, dev_max_index,
    sizeof(int), hipMemcpyDeviceToHost));

  /* Use the max index to look into the input array */
  gpuErrChk(hipMemcpy(host_maximizer, input + *host_max_index, sizeof(int), 
    hipMemcpyDeviceToHost));

  OptimizationOutput result;
  result.maximizer = *host_maximizer;
  result.maximum = *host_max;

  return result;
}

/* High level function which performs uniform random optimization */
OptimizationOutput optimize_UR(int N, float low, float high) {
  float *dev_input;

  gpuErrChk(hipMalloc(&dev_input, N * sizeof(float)));

  cout << "Generating GPU inputs" << endl;

  /* Create random inputs */
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniform(gen, dev_input, N);

  cout << "Generating GPU outputs" << endl;

  float *dev_output;
  gpuErrChk(hipMalloc(&dev_output, N * sizeof(float)));

  /* Apply the objective function to the inputs */
  callApplyFunctionKernel(dev_input, dev_output, N, low, high);

  /* Find the maximum of the outputs. */
  return cudaCallMaximumKernel(dev_input, dev_output, N);
}

OptimizationOutput optimize(int N, float low, float high, OptimizationAlgorithm algorithm) {
  switch (algorithm) {
    case UNIFORM_RANDOM:
      return optimize_UR(N, low, high);
  }

  cerr << "Unrecognized algorithm." << endl;
  throw 20;
}














